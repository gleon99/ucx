#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cuda_util.h"


/**
 * Linear congruential generator (LCG):
 * n[i + 1] = (n[i] * A + C) % M
 * where A, C, M used as in glibc
 */
    void IoDemoRandom::srand(unsigned seed) {
        _seed = seed & _M;
    }

   /*  template <typename T>
    T IoDemoRandom::rand(T min, T max) 

    template <typename T>
    T IoDemoRandom::rand(unsigned &seed, T min, T max)  */

    void IoDemoRandom::fill(unsigned &seed, void *buffer, size_t size) {
        size_t body_count = size / sizeof(uint64_t);
        size_t tail_count = size & (sizeof(uint64_t) - 1);
        uint64_t *body    = reinterpret_cast<uint64_t*>(buffer);
        uint8_t *tail     = reinterpret_cast<uint8_t*>(body + body_count);

        fill(seed, body, body_count);
        fill(seed, tail, tail_count);
    }

    size_t IoDemoRandom::validate(unsigned &seed, const void *buffer,
                                  size_t size) {
        size_t body_count    = size / sizeof(uint64_t);
        size_t tail_count    = size & (sizeof(uint64_t) - 1);
        const uint64_t *body = reinterpret_cast<const uint64_t*>(buffer);
        const uint8_t *tail  = reinterpret_cast<const uint8_t*>(body + body_count);

        size_t err_pos = validate(seed, body, body_count);
        if (err_pos < body_count) {
            return err_pos * sizeof(body[0]);
        }

        err_pos = validate(seed, tail, tail_count);
        if (err_pos < tail_count) {
            return (body_count * sizeof(body[0])) + (err_pos * sizeof(tail[0]));
        }

        return size;
    }

    void IoDemoRandom::setMemoryType(ucs_memory_type_t memory_type)
    {
        _memory_type = memory_type;
    }

    template <typename T>
    void IoDemoRandom::fill(unsigned &seed, T *buffer, size_t count) {
#ifdef HAVE_CUDA
        T temp;
#endif

        for (size_t i = 0; i < count; ++i) {
            switch (_memory_type) {
#ifdef HAVE_CUDA
            case UCS_MEMORY_TYPE_CUDA:
                temp = rand<T>(seed);
                hipMemcpy(&buffer[i], &temp, sizeof(T), hipMemcpyDefault);
                break;
            case UCS_MEMORY_TYPE_CUDA_MANAGED:
#endif
            case UCS_MEMORY_TYPE_HOST:
                buffer[i] = rand<T>(seed);
                break;
            default:
                /* Unreachable - would fail in ctor */
                abort();
            }
        }
    }

    template <typename T>
    size_t IoDemoRandom::validate(unsigned &seed, const T *buffer, size_t count) {
#ifdef HAVE_CUDA
        T expected_value, actual_value;
#endif

        for (size_t i = 0; i < count; ++i) {
            switch (_memory_type) {
#ifdef HAVE_CUDA
            case UCS_MEMORY_TYPE_CUDA:
                expected_value = rand<T>(seed);
                hipMemcpy(&actual_value, &buffer[i], sizeof(T),
                           hipMemcpyDefault);
                if (actual_value != expected_value) {
                    return i;
                }
                break;
            case UCS_MEMORY_TYPE_CUDA_MANAGED:
#endif
            case UCS_MEMORY_TYPE_HOST:
                if (buffer[i] != rand<T>(seed)) {
                    return i;
                }
                break;

            default:
                /* Unreachable - would fail in ctor */
                abort();
            }
        }

        return count;
    }

//     static       unsigned     _seed;
//     static const unsigned     _A;
//     static const unsigned     _C;
//     static const unsigned     _M;
//     static ucs_memory_type_t _memory_type;
// };
unsigned IoDemoRandom::_seed    = 0;
const unsigned IoDemoRandom::_A = 1103515245U;
const unsigned IoDemoRandom::_C = 12345U;
const unsigned IoDemoRandom::_M = 0x7fffffffU;
ucs_memory_type_t IoDemoRandom::_memory_type = UCS_MEMORY_TYPE_HOST;



const static int cuda_util_blocks_num = 256;
const static int cuda_util_threads_per_block = 256;

__global__
void LEO_add(int *x)
{
    printf("LEO %d\n", *x);
    *x += 1;
    printf("LEO %d\n", *x);
    // x += y;
}

void LEO_add2(int *x)
{
    LEO_add<<<cuda_util_blocks_num, cuda_util_threads_per_block>>>(x);
    hipDeviceSynchronize();
}
